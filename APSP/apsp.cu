#include "hip/hip_runtime.h"
 
/**************************************************
 *  All-pairs-shortest paths
 *	Host code.
 *  Recursive in-place implementation 
 *  Copyright by Ceren Budak, Aydin Buluc, Fenglin Liao, Arda Atali
 *  NOTES: Continues the recursion until all of the problem fits into one block.  
 *  DATE:  December 2007
 *  ROW-WISE (OLD) TIMINGS: [Max BLOCK_SIZE possible is 16]
	~5100/5900 ms with 4K vertices and BLOCK_SIZE=16
	~7500 ms with 4K vertices and BLOCK_SIZE=8
 *  COL-WISE (NEW) TIMINGS: [Using Volkov's GEMM]
	~1014 ms with 4K vertices
 ***************************************************/
 
 /** 
 * An implementation of the recursive APSP algorithm
 * A is an adjacency matrix of a graph, nonzeros represents edge, zeros represent no edges
 * Matrices are laid out in column-major order
 */


// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits.h>

// Project includes
#include <cutil.h>

// Kernel includes
#include "apsp_kernel.h"

using namespace std;


void runTest(int argc, char** argv);
void printDiff(float *, float*, int, int);
void floydWarshall(float *, int, int);

extern "C"
void computeGoldCol(float *, const float *, unsigned int);



int main(int argc, char** argv)
{
    runTest(argc, argv);

    //CUT_EXIT(argc, argv);
}

void Load(FILE * fid, float * distMatrix, int size)
{
	int read = 0;
	int v1, v2;
	float value;
	
	for (int j=0; j<WA; j++) 
		for (int i=0; i<WA; i++) 
			distMatrix[j*WA+i] = FLOATINF;

	while (read < size)
	{
		if(fscanf(fid, "%d\t%d\t%f\n", &v1,&v2,&value) == EOF)
		{
			fprintf(stderr,"Error reading file when reading %d of %d\n", read, size);
			exit(1);
		}
		
		

		
		distMatrix[v2*WA + v1] = fabs(value)*1000;		// column-major
		read++;
	}
	for (int i=0; i<WA; i++)
		distMatrix[i*WA + i] = 0;	// diagonals are zero
		
	fclose(fid); 
}


void runTest(int argc, char** argv)
{
    printf("Final APSP Optimized (column-major)\n");
    CUT_DEVICE_INIT(argc,argv);

 
    // allocate host memory for matrices A
    unsigned int size_A = WA * WA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float * h_A = (float*) malloc(mem_size_A);


    // Initialize host memory by reading the matrix from file
    
    FILE * fp;
    if((fp=fopen("rmat.txt", "r")) == NULL) 
    {
	printf("Cannot open file.\n");
	exit(1);
    }    int m,n, nnz;
    if(fscanf(fp, "%d\t%d\t%d\n", &m,&n,&nnz) == EOF)
    {
	fprintf(stderr,"Error reading file\n");
	exit(1);
    }
    printf("%d,%d,%d\n",m,n,nnz);
    Load(fp, h_A, nnz);

    // allocate device memory (since the algorithm is in-place, no memory required for the output)
    float * d_A;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));

    // copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );

    
    // create and start timer
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    // call our recursive function
    floydWarshall(d_A,0, WA);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
    
    // call synchronize before stopping the timer
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    
    // allocate mem for the result on host side
    float * h_C = (float*) malloc(mem_size_A);

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(h_C, d_A, mem_size_A, hipMemcpyDeviceToHost) );
    
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

#ifdef VERIFY
    // compute reference solution
   fprintf(stderr, "Computing on the CPU (for verification)\n");
   float* reference = (float*) malloc(mem_size_A);

    computeGoldCol(reference, h_A, WA);
   
    // check result
    printDiff(reference, h_C, WA, WA);
    free(reference);
#endif

    // clean up memory
    free(h_A);
    free(h_C);
    CUDA_SAFE_CALL(hipFree(d_A));
}


// recursive calls are only made to diagonal blocks, i.e. start = startx = starty

void floydWarshall(float *data, int start, int width)
{
    if(width <= BLOCK_SIZE)
    {
        // setup execution parameters
        
        // the computation now can fit in one block
        dim3 threads(width, width);
        dim3 grid(1, 1);
        
        // execute the kernel with a single block
        apsp_seq<<< grid, threads >>>(data, width,start);
    }
    else if(width <= FAST_GEMM)	
    {
	int nw = width/2;		// new width
        
        floydWarshall(data, start, nw);

        // setup execution parameters
        dim3 threadsmult(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridmult(nw / BLOCK_SIZE, nw / BLOCK_SIZE);
        
        	// execute the kernel B = AB
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start+nw, start, start,start,start+nw, start,0);
        
		// execute the kernel C = CA
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start, start+nw,start,start+nw,start, start,0);

		// execute the kernel D += CB      
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start+nw,start+nw,start,start+nw, start+nw, start,1);

		// do FW for D
	floydWarshall(data, start+nw, nw);

		// execute the kernel B = BD
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start+nw, start, start+nw,start,start+nw, start+nw,0);

		// execute the kernel C = DC
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start, start+nw,start+nw,start+nw,start, start+nw,0);

		// execute the kernel A += BC
        matrixMul<<< gridmult, threadsmult >>>(data, data, data, nw, start,start,start+nw,start, start, start+nw,1);
    }

    else
    {
        /*A=floyd-warshall(A);
        B=AB;
        C=CA;
        D=D+CB;
        D=floyd-warshall(D);
        B=BD;
        C=DC;
        A=A+BC;*/
        
        int nw = width/2;		// new width
        
        floydWarshall(data, start, nw);

        // setup execution parameters
	dim3 gemmgrid( nw/64, nw/16 );
	dim3 gemmthreads( 16, 4 );


	// Remember: Column-major
	float * A = data + start * WA + start;
	float * B = data + (start+nw) * WA + start;
	float * C = data + start * WA + (start+nw);
	float * D = data + (start+nw) * WA + (start+nw);

	// sgemmNN_MinPlus( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
	// no need to send m & n since they are known through grid dimensions !


	// execute the parallel multiplication kernel B = AB
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(A, WA, B, WA, B, WA, nw,  FLOATINF );

    	// execute the parallel multiplication kernel C = CA
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(C, WA, A, WA, C, WA, nw,  FLOATINF );
        
     
	// execute the parallel multiplication kernel  D += CB 
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(C, WA, B, WA, D, WA, nw,  1 );

	// do FW for D
	floydWarshall(data, start+nw, nw);

	// execute the parallel multiplication kernel B = BD
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(B, WA, D, WA, B, WA, nw,  FLOATINF );

	// execute the parallel multiplication kernel C = DC
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(D, WA, C, WA, C, WA, nw,  FLOATINF );

	// execute the parallel multiplication kernel A += BC
	sgemmNN_MinPlus<<<gemmgrid, gemmthreads>>>(B, WA, C, WA, A, WA, nw,  1 );
  
    }
    
}

void printDiff(float *data1, float *data2, int width, int height)
{
  fprintf(stderr,"Verifying...");

  int i,j,k;
  int error_count=0;
  for (i=0; i<height; i++) {
    for (j=0; j<width; j++) {
      k = i*width+j;
      if ( abs(data1[k] - data2[k]) > 0.01 ) {
         fprintf(stderr,"diff(%d,%d) CPU=%f, GPU=%f\n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf("\nTotal Errors = %d\n", error_count);
  
   /*	
  printf("Writing output to disk...\n"); 
 
    FILE * fp;
    if((fp=fopen("result.txt", "w")) == NULL) 
    {
		printf("Cannot open file.\n");
		exit(1);
	}
	for (int i=0; i<WA; i++) 
	{
		for (int j=0; j<WA; j++) 
		{
			if(data2[i*WA + j] != FLOATINF)
				fprintf(fp,"%d\t%d\t%f\n", i, j, data2[i*WA + j]); 
		}
	}
	
	fclose(fp);
 */
}
